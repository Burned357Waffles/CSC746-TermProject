#include "hip/hip_runtime.h"
//
// (C) 2021, E. Wes Bethel
// Created code harness for the sobel filter
// (C) 2024, Brandon Watanabe
// Modified code to be nbody simulation
//
// Usage:
//

#include <iostream>
#include <chrono>
#include <random>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <omp.h>  
#include "likwid-stuff.h"

#define DIM 3
#define HOUR 3600
#define EARTH_DAY 3600 * 24
#define EARTH_YEAR 3600 * 24 * 365

typedef double vect_t[DIM];

struct Body
{
   double mass;
   vect_t velocity;
   vect_t position;
   
   __device__ bool operator==(const Body& other) const
   {
      if (mass != other.mass)
         return false;

      for (int i = 0; i < DIM; ++i)
      {
         if (velocity[i] != other.velocity[i] || position[i] != other.position[i])
            return false;
      }

      return true;
   }
};

char output_fname[] = "../data/positions.csv";

const double G = 6.67430e-11;
const double AU = 1.496e11;
const double SOLAR_MASS = 1.989e30;
const double MERCURY_MASS = 3.285e23;
const double VENUS_MASS = 4.867e24;
const double EARTH_MASS = 5.972e24;
const double MARS_MASS = 6.39e23;
const double JUPITER_MASS = 1.898e27;
const double SATURN_MASS = 5.683e26;
const double URANUS_MASS = 8.681e25;
const double NEPTUNE_MASS = 1.024e26;
const double ASTEROID_MASS = 1.0e12;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ void 
compute_forces(Body* bodies, Body i, double* total_force, int N)
{
   for(int j = 0; j < N; j++)
   {
      if(i == bodies[j])
         continue;

      double dx[DIM] = {0.0, 0.0, 0.0};
      double r = 0.0;
      double r_norm = 0.0;

      for (int idx = 0; idx < DIM; idx++)
      {
         dx[idx] = bodies[j].position[idx] - i.position[idx];
         r += dx[idx] * dx[idx];
      }

      r_norm = sqrt(r);
      if (r_norm == 0.0)
         continue;

      for (int idx = 0; idx < DIM; idx++)
      {
         double f = (G * i.mass * bodies[j].mass * dx[idx]) / (r_norm * r_norm * r_norm);
         total_force[idx] += f;
      }
   }
}

__device__ void 
update_bodies(Body* bodies, const double* forces, const double dt, const int N, const bool record_histories, const int history_index, double* velocity_history, double* position_history)
{
   for (int i = 0; i < N; i++)
   {
      for (int idx = 0; idx < DIM; idx++)
      {
         bodies[i].velocity[idx] += forces[i * DIM + idx] / bodies[i].mass * dt;
         bodies[i].position[idx] += bodies[i].velocity[idx] * dt;
      }

      if (record_histories)
      {
         for (int idx = 0; idx < DIM; idx++)
         {
            velocity_history[history_index * N * DIM + i * DIM + idx] = bodies[i].velocity[idx];
            position_history[history_index * N * DIM + i * DIM + idx] = bodies[i].position[idx];
         }
      }
   }
}

__global__ void 
do_nBody_calculation(Body* bodies, const int N, const int timestep, const unsigned long long final_time, const bool record_histories, double* velocity_history, double* position_history, double* forces)
{
   extern __shared__ double shared_forces[];

   int history_index = 1;

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;   

   for(int t = 0; t < final_time; t+=timestep)
   {
      for (int i = threadIdx.x; i < N * DIM; i += blockDim.x)
      {
         shared_forces[i] = 0.0;
      }
      __syncthreads();

      //memset(forces, 0, N * DIM * sizeof(double));
      for (int i = index; i < N; i += stride)
      { 
         compute_forces(bodies, bodies[i], shared_forces + i * DIM, N);
      }
      __syncthreads();

      for (int i = index; i < N; i += stride)
      {
         update_bodies(bodies, shared_forces, timestep, N, record_histories, history_index, velocity_history, position_history);
      }
      __syncthreads();
      history_index++;
   }
}

void launch_nBody_calculation(Body* bodies, const int N, const int timestep, const unsigned long long final_time, const bool record_histories, double* velocity_history, double* position_history)
{
   double* forces;
   gpuErrchkcudaMallocManaged(&forces, N * DIM * sizeof(double));

   int blockSize = 256;
   int numBlocks = (N + blockSize - 1) / blockSize;
   do_nBody_calculation<<<numBlocks, blockSize>>>(bodies, N, timestep, final_time, record_histories, velocity_history, position_history, forces);
   gpuErrchk(hipGetLastError());
   gpuErrchk(hipDeviceSynchronize());

   hipFree(forces);
}

// This function will initialize the bodies with 
// random masses, initial velocities, and initial positions
// mass is in the range 1.0e-6 to SOLAR_MASS
// velocity is in the range -1.0 to 1.0
// position is in the range -1.0 to 1.0
// 
// To get an orbit:
// 1. Set the mass of the first body to SOLAR_MASS
// 2. Set the velocity of the first body to 0
// 3. Set the position of the first body to 0
// 4. Body 1 mass: 1e+12
// 5. Body 1 velocity: 22365.5, 24955.3, 28634.1
// 6/ Body 1 position: -1.77419e+09, 1.52822e+10, -2.62286e+10

Body*
init_random_bodies(const int N)
{
   Body* bodies;
   gpuErrchk(hipMallocManaged(&bodies, N * sizeof(Body)));
   std::random_device rd;
   std::mt19937 gen(rd());
   std::uniform_real_distribution<double> mass_dist(ASTEROID_MASS, SOLAR_MASS);
   std::uniform_real_distribution<double> velocity_dist(-50.0e3, 50.0e3); // Velocity in m/s
   std::uniform_real_distribution<double> position_dist(-AU, AU);

   for(int i = 0; i < N; i++)
   {
      Body body;
      body.mass = mass_dist(gen);

      for (int j = 0; j < DIM; j++)
      {
         body.velocity[j] = velocity_dist(gen);
         body.position[j] = position_dist(gen);
      }
      bodies[i] = body;
   }
   
   return bodies;
}


Body*
init_solar_system()
{
   Body* bodies;
   gpuErrchk(hipMallocManaged(&bodies, 9 * sizeof(Body)));

   double masses[] = {SOLAR_MASS, MERCURY_MASS, VENUS_MASS, EARTH_MASS, MARS_MASS, JUPITER_MASS, SATURN_MASS, URANUS_MASS, NEPTUNE_MASS};
   double velocities[][3] = {
         {0, 0, 0},
         {0, 47.87e3, 0},
         {0, 35.02e3, 0},
         {0, 29.78e3, 0},
         {0, 24.07e3, 0},
         {0, 13.07e3, 0},
         {0, 9.69e3, 0},
         {0, 6.81e3, 0},
         {0, 5.43e3, 0}
   };
   double positions[][3] = {
         {0, 0, 0},
         {0.39 * AU, 0, 0},
         {0.72 * AU, 0, 0},
         {AU, 0, 0},
         {1.52 * AU, 0, 0},
         {5.20 * AU, 0, 0},
         {9.58 * AU, 0, 0},
         {19.22 * AU, 0, 0},
         {30.05 * AU, 0, 0}
   };

   for (int i = 0; i < 9; ++i) {
      bodies[i].mass = masses[i];
      for (int j = 0; j < 3; ++j) {
         bodies[i].velocity[j] = velocities[i][j];
         bodies[i].position[j] = positions[i][j];
      }
   }

   return bodies;
}

void 
write_data_to_file(Body* bodies, const int N, const int timestep, const unsigned long long final_time, double* velocity_history, double* position_history) 
{
   int num_data_points = (final_time / timestep) + 1;

   FILE *fp = fopen(output_fname, "w");
   if (fp == NULL)
   {
      std::cerr << "Error: could not open file " << output_fname << " for writing" << std::endl;
      exit(1);
   }

   // Print header
   fprintf(fp, "body_num,m,vx,vy,vz,x,y,z\n");

   for (int i = 0; i < N; i++)
   {
      for (int j = 0; j < num_data_points; j++) {
         const double* pos = &position_history[(j * N + i) * DIM];
         const double* vel = &velocity_history[(j * N + i) * DIM];
         fprintf(fp, "%d,%f,%f,%f,%f,%f,%f,%f\n", i, bodies[i].mass, vel[0], vel[1], vel[2], pos[0], pos[1], pos[2]);
      }
   }

   fclose(fp);

   std::cout << "Data written to " << output_fname << std::endl;
}

double* allocate_history(int N, int history_length)
{
   double* history;
   gpuErrchk(hipMallocManaged(&history, history_length * N * DIM * sizeof(double)));
   return history;
}

void free_history(double* history)
{
   gpuErrchk(hipFree(history));
}

int
main (int ac, char *av[])
{
   if (ac < 7) {
      std::cerr << "Usage: " << av[0] << " <number_of_bodies> <record_histories> <timestep_modifier> <final_time_modifier> <threads_per_block> <num_blocks>" << std::endl;
      return 1;
   }

   LIKWID_MARKER_INIT;

   int N = std::stoi(av[1]);
   bool record_histories = std::stoi(av[2]);
   int timestep_modifier = std::stoi(av[3]);
   int final_time_modifier = std::stoi(av[4]);
   int threads_per_block = std::stoi(av[5]);
   int num_blocks = std::stoi(av[6]);

   int timestep = HOUR * timestep_modifier;
   unsigned long long final_time = static_cast<unsigned long long>(EARTH_YEAR) * final_time_modifier; 

   Body* bodies = nullptr;
   if (N == -1)
   {
      N = 9;
      bodies = init_solar_system();
   }
   else 
   {
      bodies = init_random_bodies(N);
   }

   int history_length = (final_time / timestep + 1);
   double* velocity_history = allocate_history(N, history_length);
   double* position_history = allocate_history(N, history_length);

   if (record_histories)
   {
      for (int i = 0; i < N; i++)
      {
         for (int idx = 0; idx < DIM; idx++)
         {
            velocity_history[i * DIM + idx] = bodies[i].velocity[idx];
            position_history[i * DIM + idx] = bodies[i].position[idx];
         }
      }
   }

   std::cout << "Number of bodies: " << N << std::endl;

   // do the processing =======================
   std::cout << "Starting nbody calculation" << std::endl;

   std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

   launch_nBody_calculation(bodies, N, timestep, final_time, record_histories, velocity_history, position_history);

   std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();

   std::chrono::duration<double> elapsed = end_time - start_time;
   std::cout << " Elapsed time is : " << elapsed.count() << " " << std::endl;

   if (record_histories)
      write_data_to_file(bodies, N, timestep, final_time, velocity_history, position_history);
   else
      std::cout << "Histories were not recorded" << std::endl;

   free_history(velocity_history);
   free_history(position_history);
   gpuErrchk(hipFree(bodies));

   LIKWID_MARKER_CLOSE;

   return 0;
}

// eof